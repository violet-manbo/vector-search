

#include <cstdint>
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <chrono>
#include <queue>
#include <sys/stat.h>    // fstat, stat, struct stat
#include <fcntl.h>       // open
#include <unordered_set>
#include <hip/hip_fp16.h>
#include <cassert>
#include <raft/core/device_mdarray.hpp>
#include <raft/core/device_resources.hpp>
#include <raft/random/make_blobs.cuh>

#include <cuvs/neighbors/cagra.hpp>

#include <rmm/mr/device/device_memory_resource.hpp>
#include <rmm/mr/device/pool_memory_resource.hpp>

#include "common.cuh"


using HostAccessor = raft::host_device_accessor<std::experimental::default_accessor<const float>, raft::memory_type::host>;
using HostMdspanT = raft::mdspan<const float, raft::matrix_extent<int64_t>, raft::row_major, HostAccessor>;



std::vector<float> read_fvecs(const std::string& filename, int64_t& num_vectors, int64_t& dim) {
  std::ifstream in(filename, std::ios::binary);
  if (!in.is_open()) {
    throw std::runtime_error("Failed to open file: " + filename);
  }

  std::vector<float> data;
  int read_dim;

  in.read(reinterpret_cast<char*>(&read_dim), sizeof(int));
  dim = read_dim;
  
 
  in.seekg(0, std::ios::beg);

  in.seekg(0, std::ios::end);
  size_t filesize = in.tellg();
  in.seekg(0, std::ios::beg);
  
  if (filesize % (dim * sizeof(float) + sizeof(int)) != 0) {
    throw std::runtime_error("Invalid fvecs file format");
  }
  
  num_vectors = filesize / (dim * sizeof(float) + sizeof(int));
  data.resize(num_vectors * dim);
  

  std::vector<float> buffer(dim);
  for (int64_t i = 0; i < num_vectors; i++) {
    in.read(reinterpret_cast<char*>(&read_dim), sizeof(int));
    if (read_dim != dim) {
      throw std::runtime_error("Inconsistent vector dimensions in file");
    }
    
    in.read(reinterpret_cast<char*>(buffer.data()), dim * sizeof(float));
    std::copy(buffer.begin(), buffer.end(), data.begin() + i * dim);
  }
  
  return data;
}



void cagra_build_and_save(raft::device_resources const& dev_resources,
                         HostMdspanT dataset,
                         const std::string& index_file,
                         int degree)
{
  using namespace cuvs::neighbors;

  //set index parameters
  cagra::index_params index_params;
  
  std::cout << "Building CAGRA index (search graph)" << std::endl;
  index_params.graph_degree = degree;
  index_params.intermediate_graph_degree = degree * 2;
  index_params.metric = cuvs::distance::DistanceType::L2Expanded;

  auto start = std::chrono::high_resolution_clock::now();
  auto index = cagra::build(dev_resources, index_params, dataset);
  std::cout << "Built index type: " << typeid(index).name() << std::endl;
  
  auto end = std::chrono::high_resolution_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
  
  std::cout << "CAGRA index built in " << duration << " ms" << std::endl;
  std::cout << "CAGRA index has " << index.size() << " vectors" << std::endl;
  std::cout << "CAGRA graph has degree " << index.graph_degree() << ", graph size ["
            << index.graph().extent(0) << ", " << index.graph().extent(1) << "]" << std::endl;
            
  // save graph
  std::cout << "Saving index to: " << index_file << std::endl;
  cagra::serialize(dev_resources, index_file, index);
  std::cout << "Index saved successfully" << std::endl;
}


void cagra_load_and_search(raft::device_resources const& dev_resources,
                          raft::device_matrix_view<const float, int64_t> queries,
                          const std::string& index_file,
                          int iterations = 10,
                          const std::string& output_file = "",
                          int search_width = 100,
                          int internal_topk = 32,
                          int team_size = 32)
{
  using namespace cuvs::neighbors;
  
  int64_t topk = 10;
  int64_t n_queries = queries.extent(0);
  
  auto neighbors = raft::make_device_matrix<uint32_t>(dev_resources, n_queries, topk);
  auto distances = raft::make_device_matrix<float>(dev_resources, n_queries, topk);
  
  
  std::cout << "Loading index from: " << index_file << std::endl;
  auto start_load = std::chrono::high_resolution_clock::now();
  
  // load graph
  cagra::index<float, unsigned int> index(dev_resources);
  cagra::deserialize(dev_resources, index_file, &index);
  auto dataset_rows = index.dataset().extent(0);
  auto dataset_dims = index.dataset().extent(1);
  printf("n_dataset: %ld, dim: %ld\n", dataset_rows, dataset_dims);
  auto end_load = std::chrono::high_resolution_clock::now();
  auto load_time = std::chrono::duration_cast<std::chrono::milliseconds>(end_load - start_load).count();
  
  std::cout << "deserialize time " << load_time << " ms" << std::endl;
  std::cout << "CAGRA index has " << index.size() << " vectors" << std::endl;
  std::cout << "CAGRA graph has degree " << index.graph_degree() << ", graph size ["
            << index.graph().extent(0) << ", " << index.graph().extent(1) << "]" << std::endl;
  
  // search parameters
  cagra::search_params search_params;
  search_params.itopk_size = internal_topk;
  search_params.search_width = search_width;
  search_params.team_size = team_size;
  search_params.max_iterations = iterations;
  search_params.algo = cagra::search_algo::SINGLE_CTA;
  search_params.persistent = false;
  std::cout << "Using search iterations: " << iterations << std::endl;
  


  cagra::search(dev_resources, search_params, index, queries, neighbors.view(), distances.view());
  

  raft::resource::sync_stream(dev_resources);

  

  if (!output_file.empty()) {
    std::cout << "Writing search results to file: " << output_file << std::endl;
    

    std::vector<uint32_t> h_neighbors(n_queries * topk);
    std::vector<float> h_distances(n_queries * topk);
    
    raft::update_host(h_neighbors.data(), neighbors.data_handle(), n_queries * topk, 
                     raft::resource::get_cuda_stream(dev_resources));
    raft::update_host(h_distances.data(), distances.data_handle(), n_queries * topk,
                     raft::resource::get_cuda_stream(dev_resources));
    raft::resource::sync_stream(dev_resources);
    

    std::ofstream outfile(output_file);
    if (!outfile) {
      std::cerr << "Error: Could not open output file: " << output_file << std::endl;
    } else {

      outfile << "# CAGRA search results" << std::endl;
      outfile << "# Queries: " << n_queries << ", K: " << topk << ", Iterations: " << iterations << std::endl;
      outfile << "# Format: neighbor_1 neighbor_2 ... neighbor_k" << std::endl;
      
      for (int64_t i = 0; i < n_queries; i++) {

        for (int64_t j = 0; j < topk; j++) {
          outfile << h_neighbors[i * topk + j];
          if (j < topk - 1) outfile << " ";
        }
        outfile << std::endl;
      }
      
      std::cout << "Results successfully written to file" << std::endl;
    }
  }
}



int main(int argc, char** argv)
{
  if (argc < 3) {
    std::cerr << "Usage:" << std::endl;
    std::cerr << "  Build index: " << argv[0] << " build <base_file.fvecs> <index_file> <degree>" << std::endl;
    std::cerr << "  Search index: " << argv[0] << " search <index_file> <query_file.fvecs> [iterations=10] [output_file] <search_width> <internal_topk> <team_size>" << std::endl;
    return 1;
  }
  
  std::string mode = argv[1];
  
  raft::device_resources dev_resources;
  
  // Set pool memory resource with 1 GiB initial pool size. All allocations use the same pool.
  rmm::mr::pool_memory_resource<rmm::mr::device_memory_resource> pool_mr(
    rmm::mr::get_current_device_resource(), 3*1024 * 1024 * 1024ull);
  rmm::mr::set_current_device_resource(&pool_mr);
  
   if (mode == "build") {
    if (argc < 4) {
      std::cerr << "Usage for build mode: " << argv[0] << " build <base_file.fvecs> <index_file>" << std::endl;
      return 1;
    }
    
    std::string base_file = argv[2];
    std::string index_file = argv[3];
    int degree = std::stoi(argv[4]);

    std::cout << "Reading base vectors from: " << base_file << std::endl;
    int64_t n_samples, n_dim;
    std::vector<float> h_base;
    try {
      h_base = read_fvecs(base_file, n_samples, n_dim);
      std::cout << "Base dataset: " << n_samples << " vectors of dimension " << n_dim << std::endl;
    } catch (const std::exception& e) {
      std::cerr << "Error reading base file: " << e.what() << std::endl;
      return 1;
    }


    HostMdspanT host_md(h_base.data(), n_samples, n_dim);

    cagra_build_and_save(dev_resources, host_md, index_file, degree);

    
  } else if (mode == "search") {
    if (argc < 4) {
      std::cerr << "Usage for search mode: " << argv[0] << " search <index_file> <query_file.fvecs> [iterations=10] [output_file] [search_width] [internal_topk] [team_size]" << std::endl;
      return 1;
    }
    
    std::string index_file = argv[2];
    std::string query_file = argv[3];
    

    int iterations = 10;
    std::string output_file = "";
    
    if (argc > 4) {

      bool is_number = true;
      for (size_t i = 0; i < strlen(argv[4]); i++) {
        if (!isdigit(argv[4][i])) {
          is_number = false;
          break;
        }
      }
      
      if (is_number) {
        iterations = std::stoi(argv[4]);

        if (argc > 5) {
          output_file = argv[5];
        }
      } else {

        output_file = argv[4];
      }
    }
    int search_width = std::stoi(argv[6]);
    int internal_topk = std::stoi(argv[7]);
    int team_size = std::stoi(argv[8]);

    std::cout << "Reading query vectors from: " << query_file << std::endl;
    int64_t n_queries, q_dim;
    std::vector<float> h_queries;
    
    try {
      h_queries = read_fvecs(query_file, n_queries, q_dim);
      std::cout << "Query dataset: " << n_queries << " vectors of dimension " << q_dim << std::endl;
    } catch (const std::exception& e) {
      std::cerr << "Error reading query file: " << e.what() << std::endl;
      return 1;
    }


    auto queries = raft::make_device_matrix<float, int64_t>(dev_resources, n_queries, q_dim);
    

    raft::update_device(queries.data_handle(), h_queries.data(), n_queries * q_dim, 
                        raft::resource::get_cuda_stream(dev_resources));
    

    raft::resource::sync_stream(dev_resources);
    
    std::cout << "Query data transferred to GPU, starting CAGRA search..." << std::endl;
    

    cagra_load_and_search(dev_resources, raft::make_const_mdspan(queries.view()), 
                         index_file, iterations, output_file, search_width, internal_topk, team_size);
    
  } else {
    std::cerr << "Unknown mode: " << mode << std::endl;
    std::cerr << "Usage:" << std::endl;
    std::cerr << "  Build index: " << argv[0] << " build <base_file.fvecs> <index_file> <degree>" << std::endl;
    std::cerr << "  Search index: " << argv[0] << " search <index_file> <query_file.fvecs> [iterations=10] [output_file]" << std::endl;
    return 1;
  }
  
  std::cout << "CAGRA processing completed." << std::endl;
  return 0;
}