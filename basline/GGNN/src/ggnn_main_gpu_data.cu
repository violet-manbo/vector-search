#include "hip/hip_runtime.h"
#include <ggnn/base/ggnn.cuh>
#include <cstddef>
#include <cstdint>
#include <iostream>
#include <fstream>
#include <string>
#include <stdexcept>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <chrono>
#include <vector>
#include <cmath>
#include <algorithm>
#include <sstream>

float* read_fvecs(const std::string& filename, size_t& num_vectors, size_t& dim) {
    std::ifstream input(filename, std::ios::binary);
    if (!input) {
        throw std::runtime_error("Unable to open .fvecs file");
    }
    int dim_int;
    input.read(reinterpret_cast<char*>(&dim_int), sizeof(int));
    dim = static_cast<size_t>(dim_int);
    input.seekg(0, std::ios::end);
    size_t filesize = input.tellg();
    input.seekg(0, std::ios::beg);
    size_t record_size = (dim + 1) * sizeof(float);
    num_vectors = filesize / record_size;
    float* data = new float[num_vectors * dim];
    for (size_t i = 0; i < num_vectors; ++i) {
        int d;
        input.read(reinterpret_cast<char*>(&d), sizeof(int));
        if (static_cast<size_t>(d) != dim) {
            throw std::runtime_error("Inconsistent vector dimension");
        }
        input.read(reinterpret_cast<char*>(data + i * dim), sizeof(float) * dim);
    }
    return data;
}

using namespace ggnn;

int main(int argc, char** argv)
{
    using GGNN = ggnn::GGNN<int32_t, float>;
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <mode (build|query)> [other arguments]" << std::endl;
        std::cerr << "Build mode: " << argv[0] << " build <base.fvecs> <graph_dir> <KBuild> <tau_build>" << std::endl;
        std::cerr << "Query mode: " << argv[0] << " query <base.fvecs> <query.fvecs> <graph_dir> <KBuild> <KQuery> <max_iterations> <distance_measure (l2|cos)> <result_file>" << std::endl;
        return 1;
    }

    std::string mode = argv[1];

    if (mode == "build") {
        if (argc != 6) {
            std::cerr << "Usage for build: " << argv[0] << " build <base.fvecs> <graph_dir> <KBuild> <tau_build>" << std::endl;
            return 1;
        }
        std::string base_path = argv[2];
        std::string graph_dir = argv[3];
        const uint32_t KBuild = std::stoi(argv[4]);
        const float tau_build = std::stof(argv[5]);
        DistanceMeasure measure = DistanceMeasure::Euclidean;
        size_t N_base, D;
        float* h_base = read_fvecs(base_path, N_base, D);
        float* base;
        hipMalloc(&base, N_base * D * sizeof(float));
        hipMemcpy(base, h_base, N_base * D * sizeof(float), hipMemcpyHostToDevice);
        GGNN ggnn{};
        int32_t gpu_id = 0;
        
        //ggnn.setShardSize(20000000u);
        ggnn.setBase(ggnn::Dataset<float>::referenceGPUData(base, N_base, D, gpu_id));
        auto start = std::chrono::high_resolution_clock::now();
        ggnn.build(KBuild, tau_build, 3, measure);
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> build_duration = end - start;
        std::cout << "Graph built in " << build_duration.count() << " ms." << std::endl;
        ggnn.setWorkingDirectory(graph_dir);
        ggnn.store();
        std::cout << "Graph built and stored successfully." << std::endl;
        hipFree(base);
        delete[] h_base;
    }
    else if (mode == "query") {
        if (argc != 10) {
            std::cerr << "Usage for query: " << argv[0] << " query <base.fvecs> <query.fvecs> <graph_dir> <KBuild> <KQuery> <max_iterations> <distance_measure (l2|cos)> <result_file>" << std::endl;
            return 1;
        }
        std::string base_path = argv[2];
        std::string query_path = argv[3];
        std::string graph_dir = argv[4];
        const uint32_t KBuild = std::stoi(argv[5]);
        const uint32_t KQuery = std::stoi(argv[6]);
        const uint32_t max_iterations = std::stoi(argv[7]);
        std::string dist_type = argv[8];
        std::string result_filename = argv[9];
        DistanceMeasure measure;
        if (dist_type == "l2") {
            measure = DistanceMeasure::Euclidean;
        } else if (dist_type == "cos") {
            measure = DistanceMeasure::Cosine;
        } else {
            std::cerr << "Invalid distance measure. Use 'l2' or 'cos'." << std::endl;
            return 1;
        }
        size_t N_base, D;
        float* h_base = read_fvecs(base_path, N_base, D);
        float* base;
        hipMalloc(&base, N_base * D * sizeof(float));
        
        GGNN ggnn{};
        int32_t gpu_id = 0;
        //ggnn.setShardSize(20000000u);
        ggnn.setWorkingDirectory(graph_dir);
        ggnn.setGPUs({0});
        ggnn.setBase(ggnn::Dataset<float>::referenceGPUData(base, N_base, D, gpu_id));
        size_t N_query;
        float* h_query = read_fvecs(query_path, N_query, D);
        float* query;
        hipMalloc(&query, N_query * D * sizeof(float));
        auto data_start = std::chrono::high_resolution_clock::now();
        hipMemcpy(base, h_base, N_base * D * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(query, h_query, N_query * D * sizeof(float), hipMemcpyHostToDevice);
        auto data_end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::micro> data_duration = data_end - data_start;
        std::cout << "Data transfer to GPU completed in " << data_duration.count()/1000.0 << " ms." << std::endl;
        ggnn::Dataset<float> d_query = ggnn::Dataset<float>::referenceGPUData(query, N_query, D, gpu_id);

        printf("Loading graph...\n");
        ggnn.load(KBuild);

        unsigned long long h_calc_count = 0;
        unsigned long long h_path_count = 0;
        unsigned long long h_fetch_time = 0;
        unsigned long long h_filter_time = 0;
        unsigned long long h_calc_time = 0;
        unsigned long long h_update_time = 0;

        printf("Start query with KBuild=%u, KQuery=%u, max_iterations=%u\n", 
               KBuild, KQuery, max_iterations);

        auto query_start = std::chrono::high_resolution_clock::now();
        const auto [indices, dists] = ggnn.query(d_query, KQuery, 0.5f, h_calc_count, h_path_count, h_fetch_time, h_filter_time, h_calc_time, h_update_time, max_iterations, measure);
        hipDeviceSynchronize();
        auto query_end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::micro> query_duration = query_end - query_start;
        std::cout << "Query completed in " << query_duration.count() << " us." << std::endl;
        std::cout << "calc count: " << h_calc_count << std::endl;
        std::cout << "path count: " << h_path_count << std::endl;
        unsigned long long sum = h_fetch_time + h_filter_time + h_calc_time + h_update_time;
        std::cout<< "fetch time ratio: " << (h_fetch_time+h_filter_time)/(double)sum << std::endl;
        std::cout<< "calc time ratio: " << h_calc_time/(double)sum << std::endl;
        std::cout<< "update time ratio: " << h_update_time/(double)sum << std::endl;

        std::ofstream result_file(result_filename);
        if (!result_file.is_open()) {
            std::cerr << "Failed to open output file!" << std::endl;
            return -1;
        }
        for (size_t i = 0; i < N_query; ++i) {
            for (int j = 0; j < KQuery; ++j) {
                result_file << indices[i * KQuery + j];
                if (j < KQuery - 1)
                    result_file << " ";
            }
            result_file << "\n";
        }
        result_file.close();
        hipFree(query);
        delete[] h_query;
        hipFree(base);
        delete[] h_base;
    }
    else {
        std::cerr << "Invalid mode. Use 'build' or 'query'." << std::endl;
        return 1;
    }
    return 0;
}